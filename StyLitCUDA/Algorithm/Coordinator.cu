#include "hip/hip_runtime.h"
#include "Coordinator.cuh"

#include "../Utilities/Image.cuh"
#include "../Utilities/Utilities.cuh"
#include "Applicator.cuh"
#include "Downscaler.cuh"
#include "NNF.cuh"
#include "PatchMatch.cuh"
#include "RandomInitializer.cuh"

#include <algorithm>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>

namespace StyLitCUDA {

template <typename T>
Coordinator<T>::Coordinator(InterfaceInput<T> &input)
    : input(input), a(input.a.rows, input.a.cols, input.a.numChannels + input.aPrime.numChannels,
                      input.numLevels),
      b(input.b.rows, input.b.cols, input.b.numChannels + input.bPrime.numChannels,
        input.numLevels),
      random(std::max(input.a.rows, input.b.rows), std::max(input.a.cols, input.b.cols), 1),
      forward(input.b.rows, input.b.cols, 1, input.numLevels),
      reverse(input.a.rows, input.a.cols, 1, input.numLevels) {
  // Loads the images into A and B.
  // A contains both A and A'.
  // B contains only B (since B' is filled in by StyLit).
  std::vector<InterfaceImage<T>> aImages(2);
  aImages[0] = input.a;
  aImages[1] = input.aPrime;
  a.levels[0].populateChannels(aImages, 0);
  std::vector<InterfaceImage<T>> bImages(1);
  bImages[0] = input.b;
  b.levels[0].populateChannels(bImages, 0);

  // Downscales the images to form the pyramid.
  for (int level = 0; level < a.levels.size() - 1; level++) {
    downscale(a.levels[level], a.levels[level + 1]);
  }
  for (int level = 0; level < b.levels.size() - 1; level++) {
    downscale(b.levels[level], b.levels[level + 1]);
  }

  // Initializes the PCG state for pseudorandom number generation.
  // Since random is passed into kernels directly, it can't use RAII.
  random.allocate();
  initializeRandomState(random);

  // Randomizes the NNFs and populates B' at the coarsest pyramid level.
  const int coarsestLevel = input.numLevels - 1;
  NNF::randomize<T>(forward.levels[coarsestLevel], random, b.levels[coarsestLevel],
                    a.levels[coarsestLevel], input.patchSize);
  NNF::randomize<T>(reverse.levels[coarsestLevel], random, a.levels[coarsestLevel],
                    b.levels[coarsestLevel], input.patchSize);
  Applicator::apply<T>(forward.levels[coarsestLevel], b.levels[coarsestLevel],
                       a.levels[coarsestLevel], input.b.numChannels,
                       input.b.numChannels + input.bPrime.numChannels, input.patchSize);

  // Runs StyLit across the pyramid, starting with the lowest level.
  for (int level = coarsestLevel; level >= 0; level--) {
    // At this stage, all NNFs and images (A, B, A', B') should be populated.
    // Improves the NNF.
    PatchMatch::run(forward.levels[level], nullptr, b.levels[level], a.levels[level], random,
                    input.patchSize, 6);

    // Upscales or applies the improved NNF, depending on the pyramid level.
    if (level) {
      NNF::upscale(forward.levels[level], forward.levels[level - 1], input.patchSize);
      Applicator::apply<T>(forward.levels[level - 1], b.levels[level - 1], a.levels[level - 1],
                           input.b.numChannels, input.b.numChannels + input.bPrime.numChannels,
                           input.patchSize);
    } else {
      // For the finest pyramid level, the NNF is applied to produce the final B'.
      Applicator::apply<T>(forward.levels[0], b.levels[0], a.levels[0], input.b.numChannels,
                           input.b.numChannels + input.bPrime.numChannels, input.patchSize);
    }
  }

  // Copies B' back to the caller.
  std::vector<InterfaceImage<T>> bImagesPrime(1);
  bImagesPrime[0] = input.bPrime;
  b.levels[0].retrieveChannels(bImagesPrime, input.b.numChannels);
}

template <typename T> Coordinator<T>::~Coordinator() { random.free(); }

template class Coordinator<int>;
template class Coordinator<float>;

void runCoordinator_float(InterfaceInput<float> &input) { Coordinator<float> coordinator(input); }

} /* namespace StyLitCUDA */
