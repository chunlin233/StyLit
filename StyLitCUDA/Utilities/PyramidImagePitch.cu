#include "PyramidImagePitch.cuh"

#include "Utilities.cuh"

#include <hip/hip_runtime.h>
#include <vector>

namespace StyLitCUDA {

template <typename T>
PyramidImagePitch<T>::PyramidImagePitch(const int rows, const int cols, const int numChannels,
                                          const int numLevels)
    : PyramidImage<T>(rows, cols, numChannels, numLevels), deviceData(nullptr),
      devicePitch(nullptr), deviceDimensions(nullptr) {}

template <typename T> void PyramidImagePitch<T>::allocate() {
  // Allocates device memory for deviceData, devicePitch and deviceDimensions.
  check(hipMalloc(&deviceData, this->numLevels * sizeof(T *)));
  check(hipMalloc(&devicePitch, this->numLevels * sizeof(size_t)));
  check(hipMalloc(&deviceDimensions, this->numLevels * sizeof(Coordinates)));

  // Populates temporary copies of deviceData, devicePitch and deviceDimensions on the host.
  std::vector<T *> hostData(this->numLevels);
  std::vector<size_t> hostPitch(this->numLevels);
  std::vector<Coordinates> hostDimensions(this->numLevels);
  hostDimensions[0] = Coordinates(this->rows, this->cols);
  for (int level = 0; level < this->numLevels; level++) {
    if (level > 0) {
      hostDimensions[level] = hostDimensions[level - 1] / 2;
    }
    check(hipMallocPitch(&hostData[level], &hostPitch[level],
                          this->numChannels * hostDimensions[level].col * sizeof(T),
                          hostDimensions[level].row));
  }

  // Copies hostData, hostPitch and hostDimensions to the device.
  check(hipMemcpy((void *) deviceData, (void *) hostData.data(), this->numLevels * sizeof(T *), hipMemcpyHostToDevice));
  check(hipMemcpy((void *) devicePitch, (void *) hostPitch.data(), this->numLevels * sizeof(size_t), hipMemcpyHostToDevice));
  check(hipMemcpy((void *) deviceDimensions, (void *) hostDimensions.data(), this->numLevels * sizeof(Coordinates), hipMemcpyHostToDevice));
}

template <typename T> void PyramidImagePitch<T>::free() {
  // Temporarily allocates memory for deviceData on the host.
  T **hostData;
  check(hipHostMalloc(&hostData, this->numLevels * sizeof(T *)));

  // Copies deviceData to the host.
  check(hipMemcpy((void *) hostData, (void *) deviceData, this->numLevels * sizeof(T *), hipMemcpyDeviceToHost));

  // Frees the images in each pyramid level.
  for (int level = 0; level < this->numLevels; level++) {
    check(hipFree((void *) hostData[level]));
  }

  // Frees deviceData, devicePitch and deviceDimensions.
  check(hipFree(deviceData));
  check(hipFree(devicePitch));
  check(hipFree(deviceDimensions));

  // Frees the temporarily allocated hostData.
  check(hipHostFree(hostData));
}

template <typename T>
__device__ const T *PyramidImagePitch<T>::at(const int row, const int col, const int level) {
  // T *image = deviceData[level];
  // T *rowStart = (T *)((char *)image + row * devicePitch[level]);
  // return &rowStart[col * this->numChannels];
  return nullptr;
}

template struct PyramidImagePitch<int>;
template struct PyramidImagePitch<float>;

} /* namespace StyLitCUDA */
