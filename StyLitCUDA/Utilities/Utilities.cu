#include "Utilities.cuh"

#include <stdio.h>

namespace StyLitCUDA {

void assertCUDA(hipError_t code, const char *file, int line, bool abort) {
  if (code != hipSuccess) {
    fprintf(stderr, "epic CUDA fail: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) {
      exit(code);
    }
  }
}

} /* namespace StyLitCUDA */
