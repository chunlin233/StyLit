#include "Vec.cuh"

#include "Utilities.cuh"

#include <hip/hip_runtime.h>

namespace StyLitCUDA {

template <typename T> Vec<T>::Vec(int size) : size(size) {}

template <typename T> void Vec<T>::deviceAllocate() {
  check(hipMalloc(&deviceData, size * sizeof(T)));
}

template <typename T> void Vec<T>::deviceFree() {
  check(hipFree(deviceData));
  deviceData = nullptr;
}

template <typename T> void Vec<T>::toDevice(T *data) {
  check(hipMemcpy(deviceData, data, size * sizeof(T), hipMemcpyHostToDevice));
}

template <typename T> void Vec<T>::fromDevice(T *data) {
  check(hipMemcpy(data, deviceData, size * sizeof(T), hipMemcpyDeviceToHost));
}

template struct Vec<float>;

} /* namespace StyLitCUDA */
